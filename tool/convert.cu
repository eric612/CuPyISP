
#include <hip/hip_runtime.h>


extern "C" __global__
void convert(const unsigned char* img,int width, int height,int bayer_pattern,unsigned char* img_out) {
    //convert RGB to Bayer
    int row = (blockIdx.y * blockDim.y + threadIdx.y)*2;
    int col = (blockIdx.x * blockDim.x + threadIdx.x)*2;
    if ((row < height) && (col < width)) {
        int offset_bayer = (row)*width + col;
        int offset_rgb = (row)*width*3 + col*3;
        if (bayer_pattern==0) {
            img_out[offset_bayer] = img[offset_rgb];
            img_out[offset_bayer+1] = img[offset_rgb+3+1];
            img_out[offset_bayer+width] = img[offset_rgb+width*3+1];
            img_out[offset_bayer+width+1] = img[offset_rgb+width*3+3+2];
        }
    }    
}
