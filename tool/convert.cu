
#include <hip/hip_runtime.h>

#define BOUND(a,min_val,max_val)           ( (a < min_val) ? min_val : (a >= max_val) ? (max_val) : a )

extern "C" __global__
void convert(const unsigned char* img,int width, int height,int bayer_pattern,unsigned short* img_out) {
    //convert RGB to Bayer
    int row = (blockIdx.y * blockDim.y + threadIdx.y)*2;
    int col = (blockIdx.x * blockDim.x + threadIdx.x)*2;
    if ((row < height) && (col < width)) {
        int offset_bayer = (row)*width + col;
        int offset_rgb = (row)*width*3 + col*3;
        if (bayer_pattern==0) {
            img_out[offset_bayer] = img[offset_rgb];
            img_out[offset_bayer+1] = img[offset_rgb+3+1];
            img_out[offset_bayer+width] = img[offset_rgb+width*3+1];
            img_out[offset_bayer+width+1] = img[offset_rgb+width*3+3+2];
        }
        else if (bayer_pattern==4) {
            float lum = (img[offset_rgb+2] + img[offset_rgb+1] + img[offset_rgb])/3;
            img_out[offset_bayer] = int(lum*16);
            offset_rgb += 3;
            lum = (img[offset_rgb+2] + img[offset_rgb+1] + img[offset_rgb])/3;
            img_out[offset_bayer+1] = int(lum*16);
            offset_rgb += (width*3 - 3);
            lum = (img[offset_rgb+2] + img[offset_rgb+1] + img[offset_rgb])/3;
            img_out[offset_bayer+width] = img[offset_rgb+2]*16;
            offset_rgb += 3;
            lum = (img[offset_rgb+2] + img[offset_rgb+1] + img[offset_rgb])/3;
            img_out[offset_bayer+width+1] = int(lum*16);
        }
    }    
}

extern "C" __global__
void convert_C_R(const unsigned char* img,int width, int height,unsigned char* img_out) {
    //convert RGB to Bayer
    int row = (blockIdx.y * blockDim.y + threadIdx.y);
    int col = (blockIdx.x * blockDim.x + threadIdx.x);
    if ((row < height) && (col < width)) {
        int offset_bayer = (row)*width + col;
        int offset_rgb = (row)*width*3 + col*3;
        int luminance = (img[offset_rgb+2] + img[offset_rgb+1] + img[offset_rgb])/3;
        img_out[offset_bayer] = abs(luminance - img[offset_rgb+2]);

    }    
}
extern "C" __global__
void Analyze_Raw(const unsigned short* img,int width, int height,unsigned short* img_out) {
    //convert RGB to Bayer
    int row = (blockIdx.y * blockDim.y + threadIdx.y)*2;
    int col = (blockIdx.x * blockDim.x + threadIdx.x)*2;
    if ((row < height-1) && (col < width-1)) {
        int offset = (row)*width + col;
        int lum = (img[offset]+img[offset+1]+img[offset+width+1])/3;
        int red = img[offset+width];
        img_out[offset] = img[offset];
        img_out[offset+1] = img[offset+1];
        img_out[offset+width] = lum;
        img_out[offset+width+1] = img[offset+width+1];
    }    
}
extern "C" __global__
void Recover_Raw(const unsigned short* img,int width, int height,int max,unsigned short* img_out) {
    //convert RGB to Bayer
    int row = (blockIdx.y * blockDim.y + threadIdx.y)*2;
    int col = (blockIdx.x * blockDim.x + threadIdx.x)*2;
    if ((row < height-1) && (col < width-1)) {
        int offset_bayer = (row)*width + col;
        int offset_rgb = (row)*width*3 + col*3;
        int lum = (img[offset_bayer]+img[offset_bayer+1]+img[offset_bayer+width+1])/3;
        int red = img[offset_bayer+width];
        int CR = lum - red;
        img_out[offset_rgb] = img[offset_bayer];
        img_out[offset_rgb+1] = img[offset_bayer];
        img_out[offset_rgb+2] = BOUND(img[offset_bayer] - CR,0,max);

        img_out[offset_rgb+3] = img[offset_bayer+1];
        img_out[offset_rgb+3+1] = img[offset_bayer+1];
        img_out[offset_rgb+3+2] = BOUND(img[offset_bayer+1] - CR,0,max);
        
        img_out[offset_rgb+width*3] = BOUND(CR+red,0,max);
        img_out[offset_rgb+width*3+1] = BOUND(CR+red,0,max);
        img_out[offset_rgb+width*3+2] = red;
        
        img_out[offset_rgb+width*3+3] = img[offset_bayer+width+1];
        img_out[offset_rgb+width*3+3+1] = img[offset_bayer+width+1];
        img_out[offset_rgb+width*3+3+2] = BOUND(img[offset_bayer+width+1] - CR,0,max);
    }    
}