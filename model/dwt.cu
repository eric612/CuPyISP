
#include <hip/hip_runtime.h>
extern "C" __global__
void dwt_h(const short* img,int width, int height,short* img_out) {
  
    int row = (blockIdx.y * blockDim.y + threadIdx.y);
    int col = (blockIdx.x * blockDim.x + threadIdx.x);
    
    if ((row < height && row>=0) && (col < width && col>=0)) {
        int offset = row*width + col;
        int offset2 = row*width + col*2;
        int offset3 = row*width + col + width/2;
        img_out[offset] = (img[offset2]+img[offset2+1]);
        img_out[offset3] = (img[offset2]-img[offset2+1]);
        
    }    
}

extern "C" __global__
void dwt_v(const short* img,int width, int height,short* img_out) {
  
    int row = (blockIdx.y * blockDim.y + threadIdx.y);
    int col = (blockIdx.x * blockDim.x + threadIdx.x);
    
    if ((row < height && row>=0) && (col < width && col>=0)) {
        int offset = row*width + col;
        int offset2 = row*width*2 + col;
        int offset3 = row*width + col + width*(height/2);
        img_out[offset] = (img[offset2]+img[offset2+width]);
        img_out[offset3] = (img[offset2]-img[offset2+width]);
        
    }    
}

extern "C" __global__
void idwt_h(const short* img,int width, int height,short* img_out) {
  
    int row = (blockIdx.y * blockDim.y + threadIdx.y);
    int col = (blockIdx.x * blockDim.x + threadIdx.x);
    
    if ((row < height && row>=0) && (col < width && col>=0)) {
        int offset = row*width + col;
        int offset2 = row*width + col*2;
        int offset3 = row*width + col + width/2;
        img_out[offset2] = (img[offset] + img[offset3])/2;
        img_out[offset2+1] = img[offset] - img_out[offset2];
        //img_out[offset] = (img[offset2]+img[offset2+1]);
        //img_out[offset3] = (img[offset2]-img[offset2+1]);
        
    }    
}

extern "C" __global__
void idwt_v(const short* img,int width, int height,short* img_out) {
  
    int row = (blockIdx.y * blockDim.y + threadIdx.y);
    int col = (blockIdx.x * blockDim.x + threadIdx.x);
    
    if ((row < height && row>=0) && (col < width && col>=0)) {
        int offset = row*width + col;
        int offset2 = row*width*2 + col;
        int offset3 = row*width + col + width*(height/2);
        img_out[offset2] = (img[offset] + img[offset3])/2;
        img_out[offset2+width] = img[offset] - img_out[offset2];
        //img_out[offset] = (img[offset2]+img[offset2+width]);
        //img_out[offset3] = (img[offset2]-img[offset2+width]);
        
    }    
}