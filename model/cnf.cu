#include "hip/hip_runtime.h"
extern "C" __global__
void cnd(const short* img, int row, int col, int width,int thres,int &is_noise,short &avgG,short &avgC1,short &avgC2){

    avgG = 0;
    avgC1 = 0;
    avgC2 = 0;
    is_noise = 0;
    for(int i=-4;i<4;i++) {
        for(int j=-4;j<4;j++) {
            int offset = (row+i)*width + col + j;
            if (i % 2 == 1 && j % 2 == 0) {
                avgG = avgG + img[offset];
            }
            else if (i % 2 == 0 && j % 2 == 1) {
                avgG = avgG + img[offset];
            }
            else if (i % 2 == 0 && j % 2 == 0) {
                avgC1 = avgC1 + img[offset];
            }
            else if (i % 2 == 1 && j % 2 == 1) {
                avgC2 = avgC2 + img[offset];
            }
        }
    }
    avgG = avgG / 40;
    avgC1 = avgC1 / 25;
    avgC2 = avgC2 / 16;
    int offset = (row+4)*width + col+4 ;
    short center = img[offset];
    if (center > (avgG) + thres && center > (avgC2) + thres) {
        if ((avgC1) > (avgG) + thres && (avgC1) > (avgC2) + thres) {
            is_noise = 1;
        }
        else {
            is_noise = 0;
        }
    }
    else {
        is_noise = 0;    
    }
}
extern "C" __global__
void cnc(int is_color, short center, short avgG, short avgC1, short avgC2,float r_gain,float b_gain,float &center_out) {
    float fade1,fade2;
    float dampFactor = 1.0;
    int signalGap = center - max(avgG, avgC2);
    float signalMeter;
    if (is_color == 0) {
        if (r_gain <= 1.0)
            dampFactor = 1.0;
        else if (r_gain > 1.0 && r_gain <= 1.2)
            dampFactor = 0.5;
        else if (r_gain > 1.2)
            dampFactor = 0.3;
    }
    else if (is_color == 2) {
        if (b_gain <= 1.0)
            dampFactor = 1.0;
        else if (b_gain > 1.0 && b_gain <= 1.2)
            dampFactor = 0.5;
        else if (b_gain > 1.2)
            dampFactor = 0.3;
    }
    float chromaCorrected = max(avgG, avgC2) + dampFactor * signalGap;
    if (is_color == 0)
        signalMeter = 0.299 * avgC1 + 0.587 * avgG + 0.114 * avgC2;
    else if (is_color == 2)
        signalMeter = 0.299 * avgC2 + 0.587 * avgG + 0.114 * avgC1;
    if (signalMeter <= 30)
        fade1 = 1.0;
    else if (signalMeter > 30 && signalMeter <= 50)
        fade1 = 0.9;
    else if (signalMeter > 50 && signalMeter <= 70)
        fade1 = 0.8;
    else if (signalMeter > 70 && signalMeter <= 100)
        fade1 = 0.7;
    else if (signalMeter > 100 && signalMeter <= 150)
        fade1 = 0.6;
    else if (signalMeter > 150 && signalMeter <= 200)
        fade1 = 0.3;
    else if (signalMeter > 200 && signalMeter <= 250)
        fade1 = 0.1;
    else
        fade1 = 0;
    if (avgC1 <= 30)
        fade2 = 1.0;
    else if (avgC1 > 30 && avgC1 <= 50)
        fade2 = 0.9;
    else if (avgC1 > 50 && avgC1 <= 70)
        fade2 = 0.8;
    else if (avgC1 > 70 && avgC1 <= 100)
        fade2 = 0.6;
    else if (avgC1 > 100 && avgC1 <= 150)
        fade2 = 0.5;
    else if (avgC1 > 150 && avgC1 <= 200)
        fade2 = 0.3;
    else if (avgC1 > 200)
        fade2 = 0;
    float fadeTot = fade1 * fade2;
    
    center_out = (1.0 - fadeTot) * center + fadeTot * chromaCorrected;
}
extern "C" __global__
void cnf_kernel(const short* img,int is_color, int row, int col,int width,int thres,float r_gain,float b_gain,short &pix_out) {
    int is_noise;
    short avgG;
    short avgC1;
    short avgC2; 
    cnd(img,row,col,width,thres,is_noise,avgG,avgC1,avgC2);
    int offset = row*width + col;
    pix_out = img[offset];
    short center = img[offset];
    float center_out;
    if (is_noise)
        cnc(is_color, center, avgG, avgC1, avgC2,r_gain,b_gain,center_out);
}
extern "C" __global__
void cnf(const short* img,int width, int height,int pad_w,int pad_h,int filter_w,int filter_h,int thres,float r_gain,float b_gain,int bayer_pattern,short* img_out) {

    int row = (blockIdx.y * blockDim.y + threadIdx.y)*2;
    int col = (blockIdx.x * blockDim.x + threadIdx.x)*2;
    int o_width = width - pad_w;
    if ((row < height) && (col < width)) {

        int offset = (row+4)*width + col + 4;
        int shift_offset = (row)*o_width + col ;
        //img_out[shift_offset] = img[offset];
        //img_out[shift_offset+1] = img[offset+1];
        //img_out[shift_offset+width] = img[offset+width];
        //img_out[shift_offset+width+1] = img[offset+width+1];
        if (bayer_pattern==0 ) {
            short r = img[offset];
            short gr = img[offset+1];
            short gb = img[offset+width];
            short b = img[offset+width+1];            
            cnf_kernel(img,0,row+4,col+4,width,thres,r_gain,b_gain,r);
            cnf_kernel(img,2,row+5,col+5,width,thres,r_gain,b_gain,b);
            img_out[shift_offset] = r;
            img_out[shift_offset+1] = gr;
            img_out[shift_offset+o_width] = gb;
            img_out[shift_offset+o_width+1] = b;                  
        }
        else if (bayer_pattern==1 ) {
            short b = img[offset];
            short gb = img[offset+1];
            short gr = img[offset+width];
            short r = img[offset+width+1];            
            cnf_kernel(img,0,row+4,col+4,width,thres,r_gain,b_gain,b);
            cnf_kernel(img,2,row+5,col+5,width,thres,r_gain,b_gain,r);
            img_out[shift_offset] = b;
            img_out[shift_offset+1] = gb;
            img_out[shift_offset+o_width] = gr;
            img_out[shift_offset+o_width+1] = r;                 
        }
        else if (bayer_pattern==2 ) {
            short gb = img[offset];
            short b = img[offset+1];
            short r = img[offset+width];
            short gr = img[offset+width+1];            
            cnf_kernel(img,0,row+4,col+5,width,thres,r_gain,b_gain,b);
            cnf_kernel(img,2,row+5,col+4,width,thres,r_gain,b_gain,r);
            img_out[shift_offset] = gb;
            img_out[shift_offset+1] = b;
            img_out[shift_offset+o_width] = r;
            img_out[shift_offset+o_width+1] = gr;             
        }
        else if (bayer_pattern==3 ) {
            short gr = img[offset];
            short r = img[offset+1];
            short b = img[offset+width];
            short gb = img[offset+width+1];            
            cnf_kernel(img,0,row+4,col+5,width,thres,r_gain,b_gain,r);
            cnf_kernel(img,2,row+5,col+4,width,thres,r_gain,b_gain,b);
            img_out[shift_offset] = gr;
            img_out[shift_offset+1] = r;
            img_out[shift_offset+o_width] = b;
            img_out[shift_offset+o_width+1] = gb;              
        }        

    }
}
