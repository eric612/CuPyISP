#include "hip/hip_runtime.h"
#define MIN(a,b) (((a)<(b))?(a):(b))
#define MAX(a,b) (((a)>(b))?(a):(b)) 
#define BOUND(a,min_val,max_val)           ( (a < min_val) ? min_val : (a >= max_val) ? (max_val) : a )
extern "C" __global__
void SortArray(int *array, int size)
{	
	int i, j, n=size, tmp;
	for (i=0; i<n-1; i++) {
		for (j=0; j<n-1-i; j++)
			if (array[j+1] < array[j]) {  
				tmp = array[j];         
				array[j] = array[j+1];
				array[j+1] = tmp;			
			}			
	}
}
extern "C" __global__
void DPC_kernel(const short* source_image,int is_color, int row, int col,int width,int thres,int nr_thres,short *pix_out) {
    short AVGPIX[4];
    int minavg;
    int maxavg;
    short p[25]; //5x5 crop_image
    int mx,mn,lu,ru,ld,rd;
    int q[13];
    for(int i=0;i<5;i++) {
        for(int j=0;j<5;j++) {
            int index = (row+i-2)*width + col + j - 2;
            p[i*5+j] = source_image[index];
        }
    }
    if (is_color == 0) {         
        //DPC
        AVGPIX[0]=(p[10]+p[14])>>1;//RL
        AVGPIX[1]=(p[2]+p[22])>>1;//UD
        AVGPIX[2]=(p[0]+p[24])>>1;//D1
        AVGPIX[3]=(p[4]+p[20])>>1;//D2
        
        //DIFPIX[0]=abs(p[10]-p[14]);//RL
        //DIFPIX[1]=abs(p[2]-p[22]);//UD
        //DIFPIX[2]=abs(p[0]-p[24]);//D1
        //DIFPIX[3]=abs(p[4]-p[20]);//D2
        
        minavg=MIN(MIN(AVGPIX[0], AVGPIX[1]), MIN(AVGPIX[2], AVGPIX[3])); 
        maxavg=MAX(MAX(AVGPIX[0], AVGPIX[1]), MAX(AVGPIX[2], AVGPIX[3])); 
                                
        q[0]=p[0];
        q[1]=p[2];
        q[2]=p[4];
        q[3]=p[10];
        q[4]=p[12];
        q[5]=p[14];
        q[6]=p[20];
        q[7]=p[22];
        q[8]=p[24];
        
        SortArray(q, 9);
        pix_out[0] = p[12];
        //if( ((p[12]-maxavg>=thres) && (q[8]-p[12]<=nr_thres && p[12]-q[7]>=thres)) )//2011/0607 start
        //{
        //    pix_out[0] = maxavg;
        //}
        //else if( (minavg-p[12]>=DpcThd2) && (q[0]==p[12] && q[1]-p[12]>=DpcThd2) )
        if( ((minavg-p[12]>=thres) && (p[12]-q[0]<=nr_thres && q[1]-p[12]>=thres)) )//2011/0607 start
        {
            pix_out[0] = minavg;
        }
    }
    else if (is_color == 1) {
        //DPC
        int INMAX, INMIN, OUTMAX, OUTMIN;
        INMAX=MAX(MAX(p[6], p[8]), MAX(p[16], p[18]));
        INMIN=MIN(MIN(p[6], p[8]), MIN(p[16], p[18]));
        OUTMAX=MAX(MAX(p[0], p[4]), MAX(p[20], p[24]));
        OUTMIN=MIN(MIN(p[0], p[4]), MIN(p[20], p[24]));
        if(abs(p[12]-INMAX)<=nr_thres || abs(INMIN-p[12])<=nr_thres)
        {
            AVGPIX[0]=(p[10]+p[14])>>1;//RL
            AVGPIX[1]=(p[2]+p[22])>>1;//UD
            AVGPIX[2]=(p[0]+p[24])>>1;//D1
            AVGPIX[3]=(p[4]+p[20])>>1;//D2					
        }
        else
        {
            AVGPIX[0]=(p[10]+p[14])>>1;//RL
            AVGPIX[1]=(p[2]+p[22])>>1;//UD
            AVGPIX[2]=(p[6]+p[18])>>1;//D1
            AVGPIX[3]=(p[8]+p[16])>>1;//D2												
        }
        minavg=MIN(MIN(AVGPIX[0], AVGPIX[1]), MIN(AVGPIX[2], AVGPIX[3])); 
        maxavg=MAX(MAX(AVGPIX[0], AVGPIX[1]), MAX(AVGPIX[2], AVGPIX[3])); 
        
        if(1)
        {
                
            //0614 modify						
            q[0]=p[2];
            q[1]=p[6];
            q[2]=p[8];
            q[3]=p[10];
            q[4]=p[12];
            q[5]=p[14];
            q[6]=p[16];
            q[7]=p[18];
            q[8]=p[22];
                
            SortArray(q, 9);
                
            lu=(2*p[6]-p[0]);
            ru=(2*p[16]-p[20]);
            ld=(2*p[18]-p[24]);
            rd=(2*p[8]-p[4]);

            mx=MAX(MAX(lu, ru), MAX(ld, rd));
            mn=MIN(MIN(lu, ru), MIN(ld, rd));
            pix_out[0] = p[12];                           
                //if( (p[12]-maxavg>=DpcThd2) && (q[8]-p[12]<=DpcNRThd && p[12]-q[6]>=DpcThd2) )
            //if( ((p[12]-maxavg>=thres && p[12]>mx) || ((p[12]-maxavg>=thres) && (q[8]-p[12]<=nr_thres && p[12]-q[6]>=thres))) )//2011/0607 start
            //{
            //    pix_out[0] = maxavg;
            //}					
            //else if( (minavg-p[12]>=DpcThd2) && (p[12]-q[0]<=DpcNRThd && q[2]-p[12]>=DpcThd2) )
            if( ((minavg-p[12]>=thres && p[12]<mn) || ((minavg-p[12]>=thres) && (p[12]-q[0]<=nr_thres && q[2]-p[12]>=thres))) )//2011/0607 start
            {
                pix_out[0] = minavg;
            }
        }       
    }
    else if (is_color == 2) {
        //DPC
        AVGPIX[0]=(p[11]+p[13])>>1;//RL
        AVGPIX[1]=(p[7]+p[17])>>1;//UD
        AVGPIX[2]=(p[0]+p[24])>>1;//D1
        AVGPIX[3]=(p[4]+p[20])>>1;//D2
        
        //DIFPIX[0]=abs(p[10]-p[14]);//RL
        //DIFPIX[1]=abs(p[2]-p[22]);//UD
        //DIFPIX[2]=abs(p[0]-p[24]);//D1
        //DIFPIX[3]=abs(p[4]-p[20]);//D2
        
        minavg=MIN(MIN(AVGPIX[0], AVGPIX[1]), MIN(AVGPIX[2], AVGPIX[3])); 
        maxavg=MAX(MAX(AVGPIX[0], AVGPIX[1]), MAX(AVGPIX[2], AVGPIX[3])); 
                                
        q[0]=p[2];
        q[1]=p[7];
        q[2]=p[10];
        q[3]=p[11];
        q[4]=p[12];
        q[5]=p[13];
        q[6]=p[14];
        q[7]=p[17];
        q[8]=p[22];
        
        SortArray(q, 9);
        pix_out[0] = p[12];
        //if( ((p[12]-maxavg>=thres) && (q[8]-p[12]<=nr_thres && p[12]-q[7]>=thres)) )//2011/0607 start
        //{
        //    pix_out[0] = maxavg;
        //}
        //else if( (minavg-p[12]>=DpcThd2) && (q[0]==p[12] && q[1]-p[12]>=DpcThd2) )
        if( ((minavg-p[12]>=thres/2) && (p[12]-q[0]<=nr_thres/2 && q[1]-p[12]>=thres/2)) )//2011/0607 start
        {
            pix_out[0] = minavg;
        } 
    }
    else if (is_color == 3) {
        //DPC
        AVGPIX[0]=(p[10]+p[14])>>1;//RL
        AVGPIX[1]=(p[2]+p[22])>>1;//UD
        AVGPIX[2]=(p[0]+p[24])>>1;//D1
        AVGPIX[3]=(p[4]+p[20])>>1;//D2
        
        //DIFPIX[0]=abs(p[10]-p[14]);//RL
        //DIFPIX[1]=abs(p[2]-p[22]);//UD
        //DIFPIX[2]=abs(p[0]-p[24]);//D1
        //DIFPIX[3]=abs(p[4]-p[20]);//D2
        
        minavg=MIN(MIN(AVGPIX[0], AVGPIX[1]), MIN(AVGPIX[2], AVGPIX[3])); 
        maxavg=MAX(MAX(AVGPIX[0], AVGPIX[1]), MAX(AVGPIX[2], AVGPIX[3])); 
                                
        q[0]=p[0];
        q[1]=p[2];
        q[2]=p[4];
        q[3]=p[10];
        q[4]=p[12];
        q[5]=p[14];
        q[6]=p[20];
        q[7]=p[22];
        q[8]=p[24];
        
        SortArray(q, 9);
        pix_out[0] = p[12];
        //if( ((p[12]-maxavg>=thres) && (q[8]-p[12]<=nr_thres && p[12]-q[7]>=thres)) )//2011/0607 start
        //{
        //    pix_out[0] = maxavg;
        //}
        //else if( (minavg-p[12]>=DpcThd2) && (q[0]==p[12] && q[1]-p[12]>=DpcThd2) )
        if( ((minavg-p[12]>=thres) && (p[12]-q[0]<=nr_thres && q[1]-p[12]>=thres)) )//2011/0607 start
        {
            pix_out[0] = minavg;
        }
    }
   
}
extern "C" __global__
void dpc(const short* src, short* dst,int width, int height,int pad_w,int pad_h,int thres,int bayer_pattern) {

    int row = (blockIdx.y * blockDim.y + threadIdx.y)*2;
    int col = (blockIdx.x * blockDim.x + threadIdx.x)*2;
    int i_width = width + pad_w;
    int i_height = height + pad_h;
    int pad_w2 = pad_w/2;
    int pad_h2 = pad_h/2;
    int offset = (row+pad_h2)*i_width + col + pad_w2;
    int shift_offset = (row)*width + col;
    int thres1 = 50;
    int thres2 = 100;
    if ((row < i_height) && (col < i_width) && row>=0 && col>=0) {
        short pix_out[3];
        DPC_kernel(src,1,row+2,col+2,i_width,thres1,thres2,pix_out);
        dst[shift_offset] = pix_out[0];
        shift_offset++;
        offset++;
        DPC_kernel(src,2,row+2,col+2+1,i_width,thres1,thres2,pix_out);
        dst[shift_offset] = pix_out[0];
        shift_offset+=width-1;
        offset+=i_width-1;
        DPC_kernel(src,0,row+2+1,col+2,i_width,thres1,thres2,pix_out);
        dst[shift_offset] = pix_out[0];
        shift_offset++;
        offset++;
        DPC_kernel(src,1,row+2+1,col+2+1,i_width,thres1,thres2,pix_out);
        dst[shift_offset] = pix_out[0];
    }
    /*
    if ((row < height) && (col < width)) {
        int offset = row*src_width + col;
        int AVGPIX[4];
        int minavg;
        int maxavg;
        short p[25]; //5x5 crop_image
        int mx,mn,lu,ru,ld,rd;
        int q[13];
        for(int i=0;i<5;i++) {
            for(int j=0;j<5;j++) {
                int offset = (row+i-2)*src_width + col + j -2;
                p[i*5+j] = src[offset];
            }
        }
        //DPC
        int INMAX, INMIN, OUTMAX, OUTMIN;
        INMAX=MAX(MAX(p[6], p[8]), MAX(p[16], p[18]));
        INMIN=MIN(MIN(p[6], p[8]), MIN(p[16], p[18]));
        OUTMAX=MAX(MAX(p[0], p[4]), MAX(p[20], p[24]));
        OUTMIN=MIN(MIN(p[0], p[4]), MIN(p[20], p[24]));
        if(abs(p[12]-INMAX)<=thres || abs(INMIN-p[12])<=thres)
        {
            AVGPIX[0]=(p[10]+p[14])>>1;//RL
            AVGPIX[1]=(p[2]+p[22])>>1;//UD
            AVGPIX[2]=(p[0]+p[24])>>1;//D1
            AVGPIX[3]=(p[4]+p[20])>>1;//D2					
        }
        else
        {
            AVGPIX[0]=(p[10]+p[14])>>1;//RL
            AVGPIX[1]=(p[2]+p[22])>>1;//UD
            AVGPIX[2]=(p[6]+p[18])>>1;//D1
            AVGPIX[3]=(p[8]+p[16])>>1;//D2												
        }
        minavg=MIN(MIN(AVGPIX[0], AVGPIX[1]), MIN(AVGPIX[2], AVGPIX[3])); 
        maxavg=MAX(MAX(AVGPIX[0], AVGPIX[1]), MAX(AVGPIX[2], AVGPIX[3])); 
        if(1) {
            //0614 modify		
            
            q[0]=p[2];
            q[1]=p[6];
            q[2]=p[8];
            q[3]=p[10];
            q[4]=p[12];
            q[5]=p[14];
            q[6]=p[16];
            q[7]=p[18];
            q[8]=p[22];
                
            SortArray(q, 9);
                
            lu=(2*p[6]-p[0]);
            ru=(2*p[16]-p[20]);
            ld=(2*p[18]-p[24]);
            rd=(2*p[8]-p[4]);

            mx=MAX(MAX(lu, ru), MAX(ld, rd));
            mn=MIN(MIN(lu, ru), MIN(ld, rd));
            dst[row*width + col] = src[offset+src_width*2+2];
            
                
                //if( (p[12]-maxavg>=DpcThd2) && (q[8]-p[12]<=DpcNRThd && p[12]-q[6]>=DpcThd2) )
            //if( ((p[12]-maxavg>=thres && p[12]>mx) || ((p[12]-maxavg>=thres) && (q[8]-p[12]<=thres && p[12]-q[6]>=thres))) )//2011/0607 start
            //{
            //    dst[row*width + col] = maxavg;
            //}					
            //else if( (minavg-p[12]>=DpcThd2) && (p[12]-q[0]<=DpcNRThd && q[2]-p[12]>=DpcThd2) )
            if( ((minavg-p[12]>=thres && p[12]<mn) || ((minavg-p[12]>=thres) && (p[12]-q[0]<=thres && q[2]-p[12]>=thres))) )//2011/0607 start
            {
                dst[row*width + col] = minavg;
            }        
        }
        if(0)
        {
            //Local Peak				
            lu=(2*p[6]-p[0]);
            ru=(2*p[16]-p[20]);
            ld=(2*p[18]-p[24]);
            rd=(2*p[8]-p[4]);
            
            mx=MAX(MAX(lu, ru), MAX(ld, rd));
            mn=MIN(MIN(lu, ru), MIN(ld, rd));
            offset = row*src_width + col;
            if( (p[12]-maxavg>=thres/2 && p[12]>mx) )
            {					
                dst[row*width + col]=maxavg;					
            }
            else if( (minavg-p[12]>=thres/2 && p[12]<mn) )
            {					
                dst[row*width + col]=minavg;					
            }
            else {
                dst[row*width + col] = src[offset+src_width*2+2];
            }
        } */
        //offset = row*src_width + col;
        //dst[row*width + col] = src[offset+src_width*2+2];
        //z[offset] = p[offset];
        /*
        int p0 = src[offset+src_width*2+2];
        int p1 = src[offset];
        int p2 = src[offset+2];
        int p3 = src[offset+4];
        int p4 = src[offset+src_width*2];
        int p5 = src[offset+src_width*2+4];
        int p6 = src[offset+src_width*4];
        int p7 = src[offset+src_width*4+2];
        int p8 = src[offset+src_width*4+4];
        if ((abs(p1 - p0) > thres) && (abs(p2 - p0) > thres) && (abs(p3 - p0) > thres) && (abs(p4 - p0) > thres) &&
        (abs(p5 - p0) > thres) && (abs(p6 - p0) > thres) && (abs(p7 - p0) > thres) && (abs(p8 - p0) > thres))
        {
                int dv = abs(2 * p0 - p2 - p7);
                int dh = abs(2 * p0 - p4 - p5);
                int ddl = abs(2 * p0 - p1 - p8);
                int ddr = abs(2 * p0 - p3 - p6);
                if (dv <= dh && dv <= ddl && dv <= ddr)
                    p0 = (p2 + p7 + 1) / 2;
                else if (dh <= dv && dh <= ddl && dh <= ddr)
                    p0 = (p4 + p5 + 1) / 2;
                else if (ddl <= dv && ddl <= dh && ddl <= ddr)
                    p0 = (p1 + p8 + 1) / 2;
                else 
                    p0 = (p3 + p6 + 1) / 2;
        }
        offset = row*width + col;
        dst[offset] = p0;
        
    }*/
 }