
#include <hip/hip_runtime.h>
extern "C" __global__
void dpc(const short* src, short* dst,int width, int height,int pad_w,int pad_h,int thres) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int src_width = width+pad_w*2;
    int src_height = height+pad_h*2;
    if ((row < height) && (col < width)) {
        int offset = row*src_width + col;
        //z[offset] = p[offset];

        int p0 = src[offset+src_width*2+2];
        int p1 = src[offset];
        int p2 = src[offset+2];
        int p3 = src[offset+4];
        int p4 = src[offset+src_width*2];
        int p5 = src[offset+src_width*2+4];
        int p6 = src[offset+src_width*4];
        int p7 = src[offset+src_width*4+2];
        int p8 = src[offset+src_width*4+4];
        if ((abs(p1 - p0) > thres) && (abs(p2 - p0) > thres) && (abs(p3 - p0) > thres) && (abs(p4 - p0) > thres) &&
        (abs(p5 - p0) > thres) && (abs(p6 - p0) > thres) && (abs(p7 - p0) > thres) && (abs(p8 - p0) > thres))
        {
                int dv = abs(2 * p0 - p2 - p7);
                int dh = abs(2 * p0 - p4 - p5);
                int ddl = abs(2 * p0 - p1 - p8);
                int ddr = abs(2 * p0 - p3 - p6);
                if (dv <= dh && dv <= ddl && dv <= ddr)
                    p0 = (p2 + p7 + 1) / 2;
                else if (dh <= dv && dh <= ddl && dh <= ddr)
                    p0 = (p4 + p5 + 1) / 2;
                else if (ddl <= dv && ddl <= dh && ddl <= ddr)
                    p0 = (p1 + p8 + 1) / 2;
                else 
                    p0 = (p3 + p6 + 1) / 2;
        }
        offset = row*width + col;
        dst[offset] = p0;
    }
 }