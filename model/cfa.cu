#include "hip/hip_runtime.h"
#define BOUND(a,min_val,max_val)           ( (a < min_val) ? min_val : (a >= max_val) ? (max_val) : a )

extern "C" __global__
void AdamsInterpolation(const short* in, int x, int y, int width, int direction, short* pix_out, short max_cut = 30) {
    if(direction == 0)
        //return BOUND(((in[(y + 1) * width + x] + in[(y - 1) * width + x])*0.5 + (in[y * width + x] * 2 - in[(y + 2) * width + x] - in[(y - 2) * width + x])*0.25),1,255);
        pix_out[0] = (in[(y + 1) * width + x] + in[(y - 1) * width + x])*0.5;// + BOUND((in[y * width + x] * 2 - in[(y + 2) * width + x] - in[(y - 2) * width + x])*0.25,-30,30);
    else 
        //return BOUND(((in[y * width + x + 1] + in[y * width + x - 1])*0.5 + (in[y * width + x] * 2 - in[y  * width + x + 2] - in[y  * width + x - 2])*0.25), 1, 255);
        pix_out[0] = (in[y * width + x + 1] + in[y * width + x - 1])*0.5;// + BOUND((in[y * width + x] * 2 - in[y  * width + x + 2] - in[y  * width + x - 2]*0.25), -30, 30);

}
extern "C" __global__
void cfa_kernel(const short* source_image,int is_color, int row, int col,int width,short *pix_out) {
    float r,g,b;
    int r2,g2,b2;
    short img[5][5]; //5x5 crop_image
    for(int i=0;i<5;i++) {
        for(int j=0;j<5;j++) {
            int offset = (row+i-2)*width + col + j -2;
            img[i][j] = source_image[offset];
        }
    }
    short out[4];
    if (is_color == 0) {
        r2 = img[2][2];
   
        g = 4 * img[2][2] - img[0][2] - img[2][0] - img[4][2] - img[2][4]
            + 2 * (img[3][2] + img[2][3] + img[1][2] + img[2][1]);
        b = 6 * img[2][2] - 3 * (img[0][2] + img[2][0] + img[4][2] + img[2][4]) / 2
            + 2 * (img[1][1] + img[1][3] + img[3][1] + img[3][3]);
        g2 = g / 8;
        b2 = b / 8;
    }
    
    else if (is_color == 1) {
        r = 5 * img[2][2] - img[2][0] - img[1][1] - img[3][1] - img[1][3] - img[3][3] - img[2][4]
            + ((img[0][2] + img[4][2]) / 2) + (4 * (img[2][1] + img[2][3]));
        g2 = img[2][2];
        b = 5 * img[2][2] - img[0][2] - img[1][1] - img[1][3] - img[4][2] - img[3][1] - img[3][3]
            + ((img[2][0] + img[2][4]) / 2) + (4 * (img[1][2] + img[3][2]));
        r2 = r / 8;
        b2 = b / 8;
    }
    else if (is_color == 2) {
        b = 5 * img[2][2] - img[2][0] - img[1][1] - img[3][1] - img[1][3] - img[3][3] - img[2][4]
            + (img[0][2] + img[4][2]) / 2 + 4 * (img[2][1] + img[2][3]);
        g2 = img[2][2];
        r = 5 * img[2][2] - img[0][2] - img[1][1] - img[1][3] - img[4][2] - img[3][1] - img[3][3]
            + (img[2][0] + img[2][4]) / 2 + 4 * (img[1][2] + img[3][2]);
        r2 = r / 8;
        b2 = b / 8;
    }
    else if (is_color == 3) {
        g = 4 * img[2][2] - img[0][2] - img[2][0] - img[4][2] - img[2][4]
            + 2 * (img[3][2] + img[2][3] + img[1][2] + img[2][1]);
        r = 6 * img[2][2] - 3 * (img[0][2] + img[2][0] + img[4][2] + img[2][4]) / 2
            + 2 * (img[1][1] + img[1][3] + img[3][1] + img[3][3]);
        b2 = img[2][2];
        r2 = r / 8;
        g2 = g / 8;
    }
    else if (is_color == 4) {
        g = img[2][3] + img[2][1] + img[1][2] + img[3][2] + img[1][1] + img[1][3] + img[3][1] + img[3][3];
        b = img[2][3] + img[2][1] + img[1][2] + img[3][2] + img[1][1] + img[1][3] + img[3][1] + img[3][3];
        r2 = img[2][2];
        int h = img[2][2]*2 - img[2][0] - img[2][4];
        int v = img[2][2]*2 - img[0][2] - img[4][2];
        if (abs(h)>abs(v)) {
            g2 = (g+v) / 8;
            b2 = (b+v) / 8;
        }
        else {
            g2 = (g+h) / 8;
            b2 = (b+h) / 8;            
        }
    }
    else if (is_color == 5) {
        AdamsInterpolation(source_image,col,row,width,1,out);
        int h = (img[2][2]*2 - img[2][0] - img[2][4])/4;
        int v = (img[2][2]*2 - img[0][2] - img[4][2])/4;
        if (abs(h)>abs(v)) {
            r = (out[0]+v);
        }
        else {
            r = (out[0]+h);          
        }        
        //r = out[0];
        g2 = img[2][2];
        b2 = img[2][2];
        r2 = r;
    }
    else if (is_color == 6) {
        AdamsInterpolation(source_image,col,row,width,0,out);
        int h = (img[2][2]*2 - img[2][0] - img[2][4])/4;
        int v = (img[2][2]*2 - img[0][2] - img[4][2])/4;
        if (abs(h)>abs(v)) {
            r = (out[0]+v);
        }
        else {
            r = (out[0]+h);          
        }
        //r = out[0];
        g2 = img[2][2];
        b2 = img[2][2];
        r2 = r;
    }
    else if (is_color == 7) {
        AdamsInterpolation(source_image,col-1,row,width,0,&out[0]);
        AdamsInterpolation(source_image,col,row-1,width,1,&out[1]);
        AdamsInterpolation(source_image,col+1,row,width,0,&out[2]);
        AdamsInterpolation(source_image,col,row+1,width,1,&out[3]);
        int h = (img[2][2]*2 - img[2][1] - img[2][3]);
        int v = (img[2][2]*2 - img[1][2] - img[3][2]);
        if (abs(h)>abs(v)) {
            r = (out[0] + out[1] + out[2] + out[3] );
        }
        else {
            r = (out[0] + out[1] + out[2] + out[3] );          
        }
        //r = out[0] + out[1] + out[2] + out[3] + img[1][1] + img[1][3] + img[3][1] + img[3][3];
        
        g2 = img[2][2];
        b2 = img[2][2];
        r2 = r/4;
    }    
    r2 = max(r2,0);
    r2 = min(r2,4095);
    g2 = max(g2,0);
    g2 = min(g2,4095);
    b2 = max(b2,0);
    b2 = min(b2,4095);
    pix_out[0] = r2;
    pix_out[1] = g2;
    pix_out[2] = b2;
    
}
extern "C" __global__
void RCCC_kernel(const short* source_image,int is_color, int row, int col,int width,short *pix_out) {
    float r,g,b;
    int r2,g2,b2;
    short img[5][5]; //5x5 crop_image
    for(int i=0;i<5;i++) {
        for(int j=0;j<5;j++) {
            int offset = (row+i-2)*width + col + j -2;
            img[i][j] = source_image[offset];
        }
    }
    short out[4];
    int lum = 0;
    int CR = 0; // lum - R
    int red = 0;
    if (is_color == 0) { 
        lum = (img[2][3]*2 + img[2][1]*2 + img[1][2]*2 + img[3][2]*2 + img[1][1] + img[1][3] + img[3][1] + img[3][3])/12;
        red = img[2][2];
        CR = lum - red;
        r2 = red;
        g2 = BOUND(CR+red,0,4095);
        b2 = BOUND(CR+red,0,4095);
    }
    else if (is_color == 1) {
        lum = (img[2][2]*6 + img[1][2] + img[3][2] + img[1][1] + img[1][3] + img[3][1] + img[3][3])/12;
        red = (img[2][1]*2 + img[2][3]*2 + img[0][1] + img[4][1] + img[0][3] + img[4][3])/8;
        /*int h = (img[2][2]*2 - img[2][0] - img[2][4])/4;
        int v = (img[2][2]*2 + img[1][2] + img[3][2] - img[1][1] - img[1][3] - img[3][1] - img[3][3]);
        if (abs(h)>abs(v)) {
            red = red + v;
        }
        else {
            red = red + h;
        }*/            
        CR = lum - red;
        r2 = red;
        g2 = BOUND(CR+red,0,4095);
        b2 = BOUND(CR+red,0,4095);        
    }
    else if (is_color == 2) {
        lum = (img[2][2]*6 + img[2][1] + img[2][3] + img[1][1] + img[1][3] + img[3][1] + img[3][3])/12;
        red = (img[1][2]*2 + img[3][2]*2 + img[1][0] + img[1][4] + img[3][0] + img[3][4])/8;

        CR = lum - red;
        r2 = red;
        g2 = BOUND(CR+red,0,4095);
        b2 = BOUND(CR+red,0,4095);  
    }
    else if (is_color == 3) {
        lum = (img[2][2]*4 + img[1][2] + img[3][2] + img[2][1] + img[2][3])/8;
        red = (img[1][1] + img[1][3] + img[3][1] + img[3][3])/4;
        CR = lum - red;
        r2 = red;
        g2 = BOUND(CR+red,0,4095);
        b2 = BOUND(CR+red,0,4095);
    }
    /*
    r2 = max(r2,0);
    r2 = min(r2,4095);
    g2 = max(g2,0);
    g2 = min(g2,4095);
    b2 = max(b2,0);
    b2 = min(b2,4095);*/
    pix_out[0] = BOUND(g2-r2*0.5,0,4095);
    pix_out[1] = g2;
    pix_out[2] = r2;    
}
extern "C" __global__
void pre_maps(const short* img,int width, int height,int pad_w,int pad_h,int bayer_pattern,short* img_out) {
    int row = (blockIdx.y * blockDim.y + threadIdx.y)*2;
    int col = (blockIdx.x * blockDim.x + threadIdx.x)*2;
    int i_width = width + pad_w;
    int i_height = height + pad_h;
    int pad_w2 = pad_w/2;
    int pad_h2 = pad_h/2;
    if ((row < i_height) && (col < i_width) && row>=0 && col>=0) {

        int offset = (row+2)*i_width + col + 2;
        int shift_offset = (row)*width*3 + col*3 ;
        short pix_out[3];
        if (bayer_pattern==4 ) {
        
        }
    }        
}
extern "C" __global__
void cfa(const short* img,int width, int height,int pad_w,int pad_h,int bayer_pattern,short* img_out) {

    int row = (blockIdx.y * blockDim.y + threadIdx.y)*2;
    int col = (blockIdx.x * blockDim.x + threadIdx.x)*2;
    int i_width = width + pad_w;
    int i_height = height + pad_h;
    int pad_w2 = pad_w/2;
    int pad_h2 = pad_h/2;
    if ((row < i_height) && (col < i_width) && row>=0 && col>=0) {

        int offset = (row+2)*i_width + col + 2;
        int shift_offset = (row)*width*3 + col*3 ;
        short pix_out[3];
        if (bayer_pattern==0 ) {
                      
            cfa_kernel(img,0,row+2,col+2,i_width,pix_out);            
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];
            
            cfa_kernel(img,1,row+2,col+3,i_width,pix_out);
            shift_offset +=3 ;
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];  
            
            cfa_kernel(img,2,row+3,col+2,i_width,pix_out);
            shift_offset += (width*3-3);
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];
            
            cfa_kernel(img,3,row+3,col+3,i_width,pix_out);
            shift_offset +=3 ;
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];             
            
        }
        else if (bayer_pattern==1 ) {           

            cfa_kernel(img,3,row+2,col+2,i_width,pix_out);            
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];
            
            cfa_kernel(img,2,row+2,col+3,i_width,pix_out);
            shift_offset +=3 ;
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];  
            
            cfa_kernel(img,1,row+3,col+2,i_width,pix_out);
            shift_offset += (width*3-3);
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];
            
            cfa_kernel(img,0,row+3,col+3,i_width,pix_out);
            shift_offset +=3 ;
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];                
        }
        else if (bayer_pattern==2 ) {          

            cfa_kernel(img,2,row+2,col+2,i_width,pix_out);            
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];
            
            cfa_kernel(img,3,row+2,col+3,i_width,pix_out);
            shift_offset +=3 ;
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];  
            
            cfa_kernel(img,0,row+3,col+2,i_width,pix_out);
            shift_offset += (width*3-3);
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];
            
            cfa_kernel(img,1,row+3,col+3,i_width,pix_out);
            shift_offset +=3 ;
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];              
        }
        else if (bayer_pattern==3 ) {          

            cfa_kernel(img,1,row+2,col+2,i_width,pix_out);            
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];
            
            cfa_kernel(img,0,row+2,col+3,i_width,pix_out);
            shift_offset +=3 ;
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];  
            
            cfa_kernel(img,3,row+3,col+2,i_width,pix_out);
            shift_offset += (width*3-3);
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];
            
            cfa_kernel(img,2,row+3,col+3,i_width,pix_out);
            shift_offset +=3 ;
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];             
        }        
        else if (bayer_pattern==4 ) {    
            RCCC_kernel(img,2,row+2,col+2,i_width,pix_out);            
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];
            
            RCCC_kernel(img,3,row+2,col+3,i_width,pix_out);
            shift_offset +=3 ;
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];  
            
            RCCC_kernel(img,0,row+3,col+2,i_width,pix_out);
            shift_offset += (width*3-3);
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];
            
            RCCC_kernel(img,1,row+3,col+3,i_width,pix_out);
            shift_offset +=3 ;
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];          
            /*
            //cfa_kernel(img,0,row+2,col+2,i_width,pix_out);   
            //int pix = (img[offset]+img[offset+1]+img[offset+i_width]+img[offset+i_width+1])/4;
            img_out[shift_offset] = img[offset];
            img_out[shift_offset+1] = img[offset];
            img_out[shift_offset+2] = img[offset];
            
            //cfa_kernel(img,1,row+2,col+3,i_width,pix_out);
            shift_offset +=3 ;
            img_out[shift_offset] = img[offset+1];
            img_out[shift_offset+1] = img[offset+1];
            img_out[shift_offset+2] = img[offset+1];  
            
            //cfa_kernel(img,2,row+3,col+2,i_width,pix_out);
            shift_offset += (width*3-3);
            img_out[shift_offset] = img[offset+i_width];
            img_out[shift_offset+1] = img[offset+i_width];
            img_out[shift_offset+2] = img[offset+i_width];
            
            //cfa_kernel(img,0,row+3,col+3,i_width,pix_out);
            shift_offset +=3 ;
            img_out[shift_offset] = img[offset+i_width+1];
            img_out[shift_offset+1] = img[offset+i_width+1];
            img_out[shift_offset+2] = img[offset+i_width+1];
            */
        } 
    }
}
