#include "hip/hip_runtime.h"
#define BOUND(a,min_val,max_val)           ( (a < min_val) ? min_val : (a >= max_val) ? (max_val) : a )

extern "C" __global__
void AdamsInterpolation(const short* in, int x, int y, int width, int direction, short* pix_out, short max_cut = 30) {
    if(direction == 0)
        //return BOUND(((in[(y + 1) * width + x] + in[(y - 1) * width + x])*0.5 + (in[y * width + x] * 2 - in[(y + 2) * width + x] - in[(y - 2) * width + x])*0.25),1,255);
        pix_out[0] = (in[(y + 1) * width + x] + in[(y - 1) * width + x])*0.5;// + BOUND((in[y * width + x] * 2 - in[(y + 2) * width + x] - in[(y - 2) * width + x])*0.25,-30,30);
    else 
        //return BOUND(((in[y * width + x + 1] + in[y * width + x - 1])*0.5 + (in[y * width + x] * 2 - in[y  * width + x + 2] - in[y  * width + x - 2])*0.25), 1, 255);
        pix_out[0] = (in[y * width + x + 1] + in[y * width + x - 1])*0.5;// + BOUND((in[y * width + x] * 2 - in[y  * width + x + 2] - in[y  * width + x - 2]*0.25), -30, 30);

}
extern "C" __global__
void cfa_kernel(const short* source_image,int is_color, int row, int col,int width,short *pix_out) {
    float r,g,b;
    int r2,g2,b2;
    short img[5][5]; //5x5 crop_image
    for(int i=0;i<5;i++) {
        for(int j=0;j<5;j++) {
            int offset = (row+i-2)*width + col + j -2;
            img[i][j] = source_image[offset];
        }
    }
    short out[4];
    if (is_color == 0) {
        r2 = img[2][2];
   
        g = 4 * img[2][2] - img[0][2] - img[2][0] - img[4][2] - img[2][4]
            + 2 * (img[3][2] + img[2][3] + img[1][2] + img[2][1]);
        b = 6 * img[2][2] - 3 * (img[0][2] + img[2][0] + img[4][2] + img[2][4]) / 2
            + 2 * (img[1][1] + img[1][3] + img[3][1] + img[3][3]);
        g2 = g / 8;
        b2 = b / 8;
    }
    
    else if (is_color == 1) {
        r = 5 * img[2][2] - img[2][0] - img[1][1] - img[3][1] - img[1][3] - img[3][3] - img[2][4]
            + ((img[0][2] + img[4][2]) / 2) + (4 * (img[2][1] + img[2][3]));
        g2 = img[2][2];
        b = 5 * img[2][2] - img[0][2] - img[1][1] - img[1][3] - img[4][2] - img[3][1] - img[3][3]
            + ((img[2][0] + img[2][4]) / 2) + (4 * (img[1][2] + img[3][2]));
        r2 = r / 8;
        b2 = b / 8;
    }
    else if (is_color == 2) {
        b = 5 * img[2][2] - img[2][0] - img[1][1] - img[3][1] - img[1][3] - img[3][3] - img[2][4]
            + (img[0][2] + img[4][2]) / 2 + 4 * (img[2][1] + img[2][3]);
        g2 = img[2][2];
        r = 5 * img[2][2] - img[0][2] - img[1][1] - img[1][3] - img[4][2] - img[3][1] - img[3][3]
            + (img[2][0] + img[2][4]) / 2 + 4 * (img[1][2] + img[3][2]);
        r2 = r / 8;
        b2 = b / 8;
    }
    else if (is_color == 3) {
        g = 4 * img[2][2] - img[0][2] - img[2][0] - img[4][2] - img[2][4]
            + 2 * (img[3][2] + img[2][3] + img[1][2] + img[2][1]);
        r = 6 * img[2][2] - 3 * (img[0][2] + img[2][0] + img[4][2] + img[2][4]) / 2
            + 2 * (img[1][1] + img[1][3] + img[3][1] + img[3][3]);
        b2 = img[2][2];
        r2 = r / 8;
        g2 = g / 8;
    }
    else if (is_color == 4) {
        g = img[2][3] + img[2][1] + img[1][2] + img[3][2] + img[1][1] + img[1][3] + img[3][1] + img[3][3];
        b = img[2][3] + img[2][1] + img[1][2] + img[3][2] + img[1][1] + img[1][3] + img[3][1] + img[3][3];
        r2 = img[2][2];
        int h = img[2][2]*2 - img[2][0] - img[2][4];
        int v = img[2][2]*2 - img[0][2] - img[4][2];
        if (abs(h)>abs(v)) {
            g2 = (g+v) / 8;
            b2 = (b+v) / 8;
        }
        else {
            g2 = (g+h) / 8;
            b2 = (b+h) / 8;            
        }
    }
    else if (is_color == 5) {
        AdamsInterpolation(source_image,col,row,width,1,out);
        int h = (img[2][2]*2 - img[2][0] - img[2][4])/4;
        int v = (img[2][2]*2 - img[0][2] - img[4][2])/4;
        if (abs(h)>abs(v)) {
            r = (out[0]+v);
        }
        else {
            r = (out[0]+h);          
        }        
        //r = out[0];
        g2 = img[2][2];
        b2 = img[2][2];
        r2 = r;
    }
    else if (is_color == 6) {
        AdamsInterpolation(source_image,col,row,width,0,out);
        int h = (img[2][2]*2 - img[2][0] - img[2][4])/4;
        int v = (img[2][2]*2 - img[0][2] - img[4][2])/4;
        if (abs(h)>abs(v)) {
            r = (out[0]+v);
        }
        else {
            r = (out[0]+h);          
        }
        //r = out[0];
        g2 = img[2][2];
        b2 = img[2][2];
        r2 = r;
    }
    else if (is_color == 7) {
        AdamsInterpolation(source_image,col-1,row,width,0,&out[0]);
        AdamsInterpolation(source_image,col,row-1,width,1,&out[1]);
        AdamsInterpolation(source_image,col+1,row,width,0,&out[2]);
        AdamsInterpolation(source_image,col,row+1,width,1,&out[3]);
        int h = (img[2][2]*2 - img[2][1] - img[2][3]);
        int v = (img[2][2]*2 - img[1][2] - img[3][2]);
        if (abs(h)>abs(v)) {
            r = (out[0] + out[1] + out[2] + out[3] );
        }
        else {
            r = (out[0] + out[1] + out[2] + out[3] );          
        }
        //r = out[0] + out[1] + out[2] + out[3] + img[1][1] + img[1][3] + img[3][1] + img[3][3];
        
        g2 = img[2][2];
        b2 = img[2][2];
        r2 = r/4;
    }    
    r2 = max(r2,0);
    r2 = min(r2,4095);
    g2 = max(g2,0);
    g2 = min(g2,4095);
    b2 = max(b2,0);
    b2 = min(b2,4095);
    pix_out[0] = r2;
    pix_out[1] = g2;
    pix_out[2] = b2;
    
}
extern "C" __global__
void Calc_CR(const short* img,int is_color, int row, int col,int width,short *pix_out) {
    int lum = 0;
    int red = 0;
    int offset = (row)*width + col ;
    if (is_color == 0) {
        int h = abs(img[offset+1]*2 - img[offset+1+width] - img[offset+1-width])+abs(img[offset-1]*2- img[offset-1-width]- img[offset-1+width])+abs((img[offset+width] - img[offset-width]));
        int v = abs(img[offset+width]*2 - img[offset+width-1] - img[offset+width+1])+abs(img[offset-width]*2 - img[offset-width-1] - img[offset-width+1]) + abs(img[offset+1] - img[offset-1]);
        int l1 = abs(img[offset+1] - img[offset-width]);
        int l2 = abs(img[offset+width] - img[offset-1]);
        int l3 = abs(img[offset+width] - img[offset+1]);
        int l4 = abs(img[offset-1] - img[offset-width]);
        int l5 = abs(img[offset+width+1] + img[offset-width-1]);
        int l6 = abs(img[offset+width-1] + img[offset-width+1]);
        
        int gradient[10],interp[10];
        gradient[0] = h;
        gradient[1] = v;
        gradient[2] = l1+l2+l5;
        gradient[3] = l3+l4+l6;
        
        
        interp[0] = (img[offset+width] + img[offset-width])/2;
        interp[1] = (img[offset+1] + img[offset-1])/2;
        interp[2] = (img[offset+width+1] + img[offset-width-1])/2;
        interp[3] = (img[offset+width-1] + img[offset-width+1])/2;
        

        int minima = 65536;
        int index = 0;
        for (int i =0;i<4;i++) {
            if(gradient[i]<minima) {
                minima = gradient[i];
                index = i;
            }
        }
        lum = interp[index];
        int avg = (interp[0]+interp[1])/2;
        int laplacian = abs(avg - img[offset+width]) + abs(avg - img[offset-width]) + abs(avg - img[offset+1]) + abs(avg - img[offset-1]);
        if(laplacian<=minima) {
            //lum = avg;
        }
        /*
        if (v<h && v<l1 && v<l2) {
            lum = (img[offset+1] + img[offset-1])/2;
        }
        else if (h<v && h<l1 && h<l2) {            
            lum = (img[offset+width] + img[offset-width])/2;
        }
        else if (l1<v && l1<h && l1<l2) {            
            lum = (img[offset+width+1] + img[offset-width-1])/2;
        }
        else {            
            lum = (img[offset-width+1] + img[offset+width-1])/2;
        }*/
        //lum = 0;
        //lum = lum = (img[offset+width] + img[offset-width])/2;
        //lum = (img[offset+width] + img[offset-width])/2;
        red = img[offset];
    }
    else if (is_color == 1) {
        lum = img[offset];
        //lum = (img[offset]*8 + img[offset-width]*2 + img[offset+width]*2 + img[offset-width-1] + img[offset-width+1] + img[offset+width-1] + img[offset+width+1])/16;
        red = (img[offset-1] + img[offset+1])/2;
    }
    else if (is_color == 2) {
        lum = img[offset];
        //lum = (img[offset]*8 + img[offset-1]*2 + img[offset+1]*2 + img[offset-width-1] + img[offset-width+1] + img[offset+width-1] + img[offset+width+1])/16;
        red = (img[offset-width] + img[offset+width])/2;
    }
    else if (is_color == 3) {
        lum = img[offset];
        //lum = (img[offset]*4 + img[offset-width] + img[offset+width] + img[offset-1] + img[offset+1])/8;
        red = (img[offset-width-1] + img[offset-width+1] + img[offset+width-1] + img[offset+width+1])/4;
        /*
        int h = abs(img[offset]*2 - img[offset-width] - img[offset+width]);
        int v = abs(img[offset]*2 - img[offset-1] - img[offset+1]);
        if (h>v) {
            //red = (img[offset+width-1] + img[offset+width+1])/2;
        }
        else {
            
            //red = (img[offset-width+1] + img[offset+width+1])/2;
        }*/        
        
    }
    pix_out[0] = lum - red;
}
extern "C" __global__
void RCCC_kernel(const short* source_image,const short* pre_maps,int is_color, int row, int col,int width,int map_width,short *pix_out) {
    float r,g,b;
    int r2,g2,b2;
    short img[5][5]; //5x5 crop_image
    short CR_img[5][5]; //5x5 crop_image
    for(int i=0;i<5;i++) {
        for(int j=0;j<5;j++) {
            int offset = (row+i-2)*width + col + j -2;
            int offset2 = (row+i-2)*map_width + col + j -2;
            img[i][j] = source_image[offset];
            CR_img[i][j] = pre_maps[offset2];
        }
    }
    short out[4];
    int lum = 0;
    int CR = 0; // lum - R
    int red = 0;
    int offset_map = (row-2)*map_width + col-2 ;
    int th = 512;
    if (is_color == 0) {         
        red = img[2][2];
        CR = pre_maps[offset_map];
        r2 = red;
        g2 = BOUND(CR+red,0,4095);
        b2 = BOUND(CR+red,0,4095);
    }
    else if (is_color == 1) {
        lum = img[2][2];
        int minimum = 9999;
        int red = (img[2][1]*2 + img[2][3]*2 + img[0][1] + img[4][1] + img[0][3] + img[4][3])/8;
        int tmp = lum - red;     
        int count = 0;
        int sum = 0;
        for(int i=0;i<5;i++) {
            for(int j=0;j<5;j++) {
                if( (i%2==0) && (j%2==1)) {
                    count++;
                    sum += CR_img[i][j];
                }
                else if (abs(CR_img[i][j] - CR_img[2][2])<th) {
                    count++;
                    sum += CR_img[i][j];                    
                }
            }
        }
        if(count!=0) {
            tmp = sum/count;
        }
        //red = (img[2][1]*2 + img[2][3]*2 + img[0][1] + img[4][1] + img[0][3] + img[4][3])/8;                  
        r2 = BOUND(lum - tmp,0,4095);
        g2 = BOUND(lum,0,4095);
        b2 = BOUND(lum,0,4095);        
    }
    else if (is_color == 2) {
        lum = img[2][2];
        int minimum = 9999;
        int red = (img[1][2]*2 + img[3][2]*2 + img[1][0] + img[1][4] + img[3][0] + img[3][4])/8; 
        int tmp = lum - red;   
        int count = 0;
        int sum = 0;        
        for(int i=0;i<5;i++) {
            for(int j=0;j<5;j++) {
                if( (i%2==1) && (j%2==0)) {
                    count++;
                    sum += CR_img[i][j];
                }
                else if (abs(CR_img[i][j] - CR_img[2][2])<th) {
                    count++;
                    sum += CR_img[i][j];                    
                }                
            }
        }
        if(count!=0) {
            tmp = sum/count;
        }
        //red = (img[1][2]*2 + img[3][2]*2 + img[1][0] + img[1][4] + img[3][0] + img[3][4])/8;
        r2 = BOUND(lum - tmp,0,4095);
        g2 = BOUND(lum,0,4095);
        b2 = BOUND(lum,0,4095);  
    }
    else if (is_color == 3) {
        lum = img[2][2];
        int minimum = 9999;
        int red = (img[1][1] + img[1][3] + img[3][1] + img[3][3])/4;
        int tmp = lum - red;  
        int count = 0;
        int sum = 0;        
        for(int i=0;i<5;i++) {
            for(int j=0;j<5;j++) {
                if( (i%2==1) && (j%2==1)) {
                    count++;
                    sum += CR_img[i][j];
                }
                else if (abs(CR_img[i][j] - CR_img[2][2])<th) {
                    count++;
                    sum += CR_img[i][j];                    
                }                
            }
        }
        if(count!=0) {
            tmp = sum/count;
        }        
        //red = (img[1][1] + img[1][3] + img[3][1] + img[3][3])/4;        
        r2 = BOUND(lum - tmp,0,4095);
        g2 = BOUND(lum,0,4095);
        b2 = BOUND(lum,0,4095);
    }
    /*
    r2 = max(r2,0);
    r2 = min(r2,4095);
    g2 = max(g2,0);
    g2 = min(g2,4095);
    b2 = max(b2,0);
    b2 = min(b2,4095);*/
    //b2 = (0.413*g2-0.299*r2)/0.114;
    //g2 = (g2 - 0.299*r2 - 0.114*b2)/0.587;
    pix_out[0] = BOUND(b2,0,4095);
    pix_out[1] = BOUND(g2,0,4095);
    pix_out[2] = BOUND(r2,0,4095);
}
extern "C" __global__
void pre_maps(const short* img,int width, int height,int pad_w,int pad_h,int bayer_pattern,short* img_out) {
    int row = (blockIdx.y * blockDim.y + threadIdx.y)*2;
    int col = (blockIdx.x * blockDim.x + threadIdx.x)*2;
    int i_width = width + pad_w;
    int i_height = height + pad_h;
    int pad_w2 = pad_w/2;
    int pad_h2 = pad_h/2;
    if ((row < i_height) && (col < i_width) && row>=0 && col>=0) {

        int offset = (row+2)*i_width + col + 2;
        int shift_offset = (row)*width + col ;
        short pix_out[3];
        if (bayer_pattern==4 ) {
            Calc_CR(img,2,row+2,col+2,i_width,pix_out);            
            img_out[shift_offset] = (pix_out[0]);
            
            Calc_CR(img,3,row+2,col+3,i_width,pix_out);
            shift_offset +=1 ;
            img_out[shift_offset] = (pix_out[0]);
            
            Calc_CR(img,0,row+3,col+2,i_width,pix_out);
            shift_offset += (width-1);
            img_out[shift_offset] = (pix_out[0]);
            
            Calc_CR(img,1,row+3,col+3,i_width,pix_out);
            shift_offset +=1 ;
            img_out[shift_offset] = (pix_out[0]);              
        }
    }        
}
extern "C" __global__
void cfa(const short* img,const short* pre_maps,int width, int height,int pad_w,int pad_h,int bayer_pattern,short* img_out) {

    int row = (blockIdx.y * blockDim.y + threadIdx.y)*2;
    int col = (blockIdx.x * blockDim.x + threadIdx.x)*2;
    int i_width = width + pad_w;
    int i_height = height + pad_h;
    int pad_w2 = pad_w/2;
    int pad_h2 = pad_h/2;
    if ((row < i_height) && (col < i_width) && row>=0 && col>=0) {

        int offset = (row+2)*i_width + col + 2;
        int shift_offset = (row)*width*3 + col*3 ;
        short pix_out[3];
        if (bayer_pattern==0 ) {
                      
            cfa_kernel(img,0,row+2,col+2,i_width,pix_out);            
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];
            
            cfa_kernel(img,1,row+2,col+3,i_width,pix_out);
            shift_offset +=3 ;
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];  
            
            cfa_kernel(img,2,row+3,col+2,i_width,pix_out);
            shift_offset += (width*3-3);
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];
            
            cfa_kernel(img,3,row+3,col+3,i_width,pix_out);
            shift_offset +=3 ;
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];             
            
        }
        else if (bayer_pattern==1 ) {           

            cfa_kernel(img,3,row+2,col+2,i_width,pix_out);            
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];
            
            cfa_kernel(img,2,row+2,col+3,i_width,pix_out);
            shift_offset +=3 ;
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];  
            
            cfa_kernel(img,1,row+3,col+2,i_width,pix_out);
            shift_offset += (width*3-3);
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];
            
            cfa_kernel(img,0,row+3,col+3,i_width,pix_out);
            shift_offset +=3 ;
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];                
        }
        else if (bayer_pattern==2 ) {          

            cfa_kernel(img,2,row+2,col+2,i_width,pix_out);            
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];
            
            cfa_kernel(img,3,row+2,col+3,i_width,pix_out);
            shift_offset +=3 ;
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];  
            
            cfa_kernel(img,0,row+3,col+2,i_width,pix_out);
            shift_offset += (width*3-3);
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];
            
            cfa_kernel(img,1,row+3,col+3,i_width,pix_out);
            shift_offset +=3 ;
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];              
        }
        else if (bayer_pattern==3 ) {          

            cfa_kernel(img,1,row+2,col+2,i_width,pix_out);            
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];
            
            cfa_kernel(img,0,row+2,col+3,i_width,pix_out);
            shift_offset +=3 ;
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];  
            
            cfa_kernel(img,3,row+3,col+2,i_width,pix_out);
            shift_offset += (width*3-3);
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];
            
            cfa_kernel(img,2,row+3,col+3,i_width,pix_out);
            shift_offset +=3 ;
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];             
        }        
        else if (bayer_pattern==4 ) {    
            RCCC_kernel(img,pre_maps,2,row+2,col+2,i_width,width,pix_out);            
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];
            
            RCCC_kernel(img,pre_maps,3,row+2,col+3,i_width,width,pix_out);
            shift_offset +=3 ;
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];  
            
            RCCC_kernel(img,pre_maps,0,row+3,col+2,i_width,width,pix_out);
            shift_offset += (width*3-3);
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];
            
            RCCC_kernel(img,pre_maps,1,row+3,col+3,i_width,width,pix_out);
            shift_offset +=3 ;
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];          
            /*
            //cfa_kernel(img,0,row+2,col+2,i_width,pix_out);   
            //int pix = (img[offset]+img[offset+1]+img[offset+i_width]+img[offset+i_width+1])/4;
            img_out[shift_offset] = img[offset];
            img_out[shift_offset+1] = img[offset];
            img_out[shift_offset+2] = img[offset];
            
            //cfa_kernel(img,1,row+2,col+3,i_width,pix_out);
            shift_offset +=3 ;
            img_out[shift_offset] = img[offset+1];
            img_out[shift_offset+1] = img[offset+1];
            img_out[shift_offset+2] = img[offset+1];  
            
            //cfa_kernel(img,2,row+3,col+2,i_width,pix_out);
            shift_offset += (width*3-3);
            img_out[shift_offset] = img[offset+i_width];
            img_out[shift_offset+1] = img[offset+i_width];
            img_out[shift_offset+2] = img[offset+i_width];
            
            //cfa_kernel(img,0,row+3,col+3,i_width,pix_out);
            shift_offset +=3 ;
            img_out[shift_offset] = img[offset+i_width+1];
            img_out[shift_offset+1] = img[offset+i_width+1];
            img_out[shift_offset+2] = img[offset+i_width+1];
            */
        } 
    }
}
