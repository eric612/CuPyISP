#include "hip/hip_runtime.h"
#define BOUND(a,min_val,max_val)           ( (a < min_val) ? min_val : (a >= max_val) ? (max_val) : a )

extern "C" __global__
void AdamsInterpolation(const short* in, int x, int y, int width, int direction, short* pix_out, short max_cut = 30) {
    if(direction == 0)
        //return BOUND(((in[(y + 1) * width + x] + in[(y - 1) * width + x])*0.5 + (in[y * width + x] * 2 - in[(y + 2) * width + x] - in[(y - 2) * width + x])*0.25),1,255);
        pix_out[0] = ((in[(y + 1) * width + x] + in[(y - 1) * width + x])*0.5 + BOUND((in[y * width + x] * 2 - in[(y + 2) * width + x] - in[(y - 2) * width + x]),-max_cut, max_cut)/4);
    else 
        //return BOUND(((in[y * width + x + 1] + in[y * width + x - 1])*0.5 + (in[y * width + x] * 2 - in[y  * width + x + 2] - in[y  * width + x - 2])*0.25), 1, 255);
        pix_out[0] = ((in[y * width + x + 1] + in[y * width + x - 1])*0.5 + BOUND((in[y * width + x] * 2 - in[y  * width + x + 2] - in[y  * width + x - 2]), -max_cut, max_cut)/4);

}
extern "C" __global__
void cfa_kernel(const short* source_image,int is_color, int row, int col,int width,short *pix_out) {
    float r,g,b;
    int r2,g2,b2;
    short img[5][5]; //5x5 crop_image
    for(int i=0;i<5;i++) {
        for(int j=0;j<5;j++) {
            int offset = (row+i-2)*width + col + j -2;
            img[i][j] = source_image[offset];
        }
    }
    short out[4];
    if (is_color == 0) {
        r2 = img[2][2];
   
        g = 4 * img[2][2] - img[0][2] - img[2][0] - img[4][2] - img[2][4]
            + 2 * (img[3][2] + img[2][3] + img[1][2] + img[2][1]);
        b = 6 * img[2][2] - 3 * (img[0][2] + img[2][0] + img[4][2] + img[2][4]) / 2
            + 2 * (img[1][1] + img[1][3] + img[3][1] + img[3][3]);
        g2 = g / 8;
        b2 = b / 8;
    }
    
    else if (is_color == 1) {
        r = 5 * img[2][2] - img[2][0] - img[1][1] - img[3][1] - img[1][3] - img[3][3] - img[2][4]
            + ((img[0][2] + img[4][2]) / 2) + (4 * (img[2][1] + img[2][3]));
        g2 = img[2][2];
        b = 5 * img[2][2] - img[0][2] - img[1][1] - img[1][3] - img[4][2] - img[3][1] - img[3][3]
            + ((img[2][0] + img[2][4]) / 2) + (4 * (img[1][2] + img[3][2]));
        r2 = r / 8;
        b2 = b / 8;
    }
    else if (is_color == 2) {
        b = 5 * img[2][2] - img[2][0] - img[1][1] - img[3][1] - img[1][3] - img[3][3] - img[2][4]
            + (img[0][2] + img[4][2]) / 2 + 4 * (img[2][1] + img[2][3]);
        g2 = img[2][2];
        r = 5 * img[2][2] - img[0][2] - img[1][1] - img[1][3] - img[4][2] - img[3][1] - img[3][3]
            + (img[2][0] + img[2][4]) / 2 + 4 * (img[1][2] + img[3][2]);
        r2 = r / 8;
        b2 = b / 8;
    }
    else if (is_color == 3) {
        g = 4 * img[2][2] - img[0][2] - img[2][0] - img[4][2] - img[2][4]
            + 2 * (img[3][2] + img[2][3] + img[1][2] + img[2][1]);
        r = 6 * img[2][2] - 3 * (img[0][2] + img[2][0] + img[4][2] + img[2][4]) / 2
            + 2 * (img[1][1] + img[1][3] + img[3][1] + img[3][3]);
        b2 = img[2][2];
        r2 = r / 8;
        g2 = g / 8;
    }
    else if (is_color == 4) {
        g = img[2][3] + img[2][1] + img[1][2] + img[3][2] + img[1][1] + img[1][3] + img[3][1] + img[3][3];
        b = img[2][3] + img[2][1] + img[1][2] + img[3][2] + img[1][1] + img[1][3] + img[3][1] + img[3][3];
        r2 = img[2][2];
        g2 = g / 8;
        b2 = b / 8;
    }
    else if (is_color == 5) {
        AdamsInterpolation(source_image,col,row,width,1,out);
        r = out[0];
        g2 = img[2][2];
        b2 = img[2][2];
        r2 = r;
    }
    else if (is_color == 6) {
        AdamsInterpolation(source_image,col,row,width,0,out);
        r = out[0];
        g2 = img[2][2];
        b2 = img[2][2];
        r2 = r;
    }
    else if (is_color == 7) {
        AdamsInterpolation(source_image,col-1,row,width,0,&out[0]);
        AdamsInterpolation(source_image,col,row-1,width,1,&out[1]);
        AdamsInterpolation(source_image,col+1,row,width,0,&out[2]);
        AdamsInterpolation(source_image,col,row+1,width,1,&out[3]);
        //r = out[0] + out[1] + out[2] + out[3] + img[1][1] + img[1][3] + img[3][1] + img[3][3];
        r = out[0] + out[1] + out[2] + out[3];
        g2 = img[2][2];
        b2 = img[2][2];
        r2 = r/4;
    }    
    r2 = max(r2,0);
    r2 = min(r2,255*4);
    g2 = max(g2,0);
    g2 = min(g2,255*4);
    b2 = max(b2,0);
    b2 = min(b2,255*4);
    pix_out[0] = r2;
    pix_out[1] = g2;
    pix_out[2] = b2;
    
}
extern "C" __global__
void cfa(const short* img,int width, int height,int pad_w,int pad_h,int bayer_pattern,short* img_out) {

    int row = (blockIdx.y * blockDim.y + threadIdx.y)*2;
    int col = (blockIdx.x * blockDim.x + threadIdx.x)*2;
    int i_width = width + pad_w;
    int i_height = height + pad_h;
    int pad_w2 = pad_w/2;
    int pad_h2 = pad_h/2;
    if ((row < i_height) && (col < i_width) && row>=0 && col>=0) {

        int offset = (row+2)*i_width + col + 2;
        int shift_offset = (row)*width*3 + col*3 ;
        short pix_out[3];
        if (bayer_pattern==0 ) {
                      
            cfa_kernel(img,0,row+2,col+2,i_width,pix_out);            
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];
            
            cfa_kernel(img,1,row+2,col+3,i_width,pix_out);
            shift_offset +=3 ;
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];  
            
            cfa_kernel(img,2,row+3,col+2,i_width,pix_out);
            shift_offset += (width*3-3);
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];
            
            cfa_kernel(img,3,row+3,col+3,i_width,pix_out);
            shift_offset +=3 ;
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];             
            
        }
        else if (bayer_pattern==1 ) {           

            cfa_kernel(img,3,row+2,col+2,i_width,pix_out);            
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];
            
            cfa_kernel(img,2,row+2,col+3,i_width,pix_out);
            shift_offset +=3 ;
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];  
            
            cfa_kernel(img,1,row+3,col+2,i_width,pix_out);
            shift_offset += (width*3-3);
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];
            
            cfa_kernel(img,0,row+3,col+3,i_width,pix_out);
            shift_offset +=3 ;
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];                
        }
        else if (bayer_pattern==2 ) {          

            cfa_kernel(img,2,row+2,col+2,i_width,pix_out);            
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];
            
            cfa_kernel(img,3,row+2,col+3,i_width,pix_out);
            shift_offset +=3 ;
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];  
            
            cfa_kernel(img,0,row+3,col+2,i_width,pix_out);
            shift_offset += (width*3-3);
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];
            
            cfa_kernel(img,1,row+3,col+3,i_width,pix_out);
            shift_offset +=3 ;
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];              
        }
        else if (bayer_pattern==3 ) {          

            cfa_kernel(img,1,row+2,col+2,i_width,pix_out);            
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];
            
            cfa_kernel(img,0,row+2,col+3,i_width,pix_out);
            shift_offset +=3 ;
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];  
            
            cfa_kernel(img,3,row+3,col+2,i_width,pix_out);
            shift_offset += (width*3-3);
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];
            
            cfa_kernel(img,2,row+3,col+3,i_width,pix_out);
            shift_offset +=3 ;
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];             
        }        
        else if (bayer_pattern==4 ) {    
            cfa_kernel(img,6,row+2,col+2,i_width,pix_out);            
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];
            
            cfa_kernel(img,7,row+2,col+3,i_width,pix_out);
            shift_offset +=3 ;
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];  
            
            cfa_kernel(img,4,row+3,col+2,i_width,pix_out);
            shift_offset += (width*3-3);
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];
            
            cfa_kernel(img,5,row+3,col+3,i_width,pix_out);
            shift_offset +=3 ;
            img_out[shift_offset] = pix_out[0];
            img_out[shift_offset+1] = pix_out[1];
            img_out[shift_offset+2] = pix_out[2];          
            /*
            //cfa_kernel(img,0,row+2,col+2,i_width,pix_out);   
            //int pix = (img[offset]+img[offset+1]+img[offset+i_width]+img[offset+i_width+1])/4;
            img_out[shift_offset] = img[offset];
            img_out[shift_offset+1] = img[offset];
            img_out[shift_offset+2] = img[offset];
            
            //cfa_kernel(img,1,row+2,col+3,i_width,pix_out);
            shift_offset +=3 ;
            img_out[shift_offset] = img[offset+1];
            img_out[shift_offset+1] = img[offset+1];
            img_out[shift_offset+2] = img[offset+1];  
            
            //cfa_kernel(img,2,row+3,col+2,i_width,pix_out);
            shift_offset += (width*3-3);
            img_out[shift_offset] = img[offset+i_width];
            img_out[shift_offset+1] = img[offset+i_width];
            img_out[shift_offset+2] = img[offset+i_width];
            
            //cfa_kernel(img,0,row+3,col+3,i_width,pix_out);
            shift_offset +=3 ;
            img_out[shift_offset] = img[offset+i_width+1];
            img_out[shift_offset+1] = img[offset+i_width+1];
            img_out[shift_offset+2] = img[offset+i_width+1];
            */
        } 
    }
}
