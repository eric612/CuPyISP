#include "hip/hip_runtime.h"
extern "C" __global__
void average(const short* array,int size,int &out) {
    int sum = 0;
    for(int i=0;i<size;i++) {
        sum = sum + array[i];
    }
    out = sum/size;
}
extern "C" __global__
void average2(const int* array,int size,int &out) {
    int sum = 0;
    for(int i=0;i<size;i++) {
        sum = sum + array[i];
    }
    out = sum/size;
}
extern "C" __global__
void subtract_power(const short* array1,const short* array2,int size,int *out) {

    for(int i=0;i<size;i++) {
        out[i] = (array1[i] - array2[i])*(array1[i] - array2[i]);
    }
}
extern "C" __global__
void weighted_sum(const short* array,const short* weight,int size,int &sum) {
    sum = 0;
    for(int i=0;i<size;i++) {
        sum = sum + weight[i]*array[i];
    }
}
extern "C" __global__
void patch(const short* source_image,int filter_size, int row, int col,int width,short *patch) {
    int fw = filter_size;
    int shift = fw/2;
    for(int i=0;i<fw;i++) {
        for(int j=0;j<fw;j++) {
            int offset = (row+i-shift)*width + col + j - shift;
            patch[i*fw+j] = source_image[offset];
        }
    }       
}
extern "C" __global__
void nlm_kernel(const short* source_image, int row, int col,int distance,int filter_size,int width,int h,short &pix_out) {
    int offset = distance/2;
    short center[9]; //3x3 crop_image
    short out[9]; //3x3 crop_image
    int sub[9];
    float average = 0;
    float weight = 0;
    float max_weight = 0;
    float wmax;
    patch(source_image,filter_size,row,col,width,center);
    
    for(int i=0;i<distance;i++) {
        for(int j=0;j<distance;j++) {
            int avg;
            int y = row + i - offset;
            int x = col + j - offset;
            //int offset = (y)*width + x;           
            patch(source_image,filter_size,y,x,width,out);
            subtract_power(out,center,9,sub);
            average2(sub,9,avg);
            float w = exp(-avg/100.0);   
            if (w > wmax)
                wmax = w;
            weight = weight + w;
            average = average + w * source_image[y*width+x];            
        }
    }
    average = average + wmax*source_image[row*width+col];
    weight = weight + wmax;
    pix_out = short(average / weight);
    
}

extern "C" __global__
void nlm(const short* img,int width, int height,int pad_w,int pad_h,int ds,int ks,int h,short* img_out) {
    // Non-local mean 
    // ks : kernel size
    // ds : search distance
    
    int row = (blockIdx.y * blockDim.y + threadIdx.y);
    int col = (blockIdx.x * blockDim.x + threadIdx.x);
    int i_width = width + pad_w;
    int i_height = height + pad_h;
    int pad_w2 = pad_w/2;
    int pad_h2 = pad_h/2;
    int avg;
    short out[9]; //3x3 crop_image
    short pix_out;
    if ((row < height && row>=0) && (col < width && col>=0)) {
        int offset = (row+pad_w2)*i_width + col + pad_w2;
        int shift_offset = (row)*width + col ;
        nlm_kernel(img,row+pad_h2,col+pad_w2,ds,ks,i_width,h,pix_out);
        //patch(img,3,row+pad_h2,col+pad_w2,i_width,out);
        //average(out,9,avg);
        img_out[shift_offset] = pix_out;
        /*
        nlm_kernel(img,row+pad_h2,col+pad_w2+1,ds,ks,i_width,h,pix_out);
        img_out[shift_offset+1] = pix_out;
        nlm_kernel(img,row+pad_h2+1,col+pad_w2,ds,ks,i_width,h,pix_out);
        img_out[shift_offset+width] = pix_out;
        nlm_kernel(img,row+pad_h2+1,col+pad_w2+1,ds,ks,i_width,h,pix_out);
        img_out[shift_offset+width+1] = pix_out;*/
        /*patch(img,3,row+pad_h2,col+pad_w2,i_width,out);
        average(out,9,avg);
        img_out[shift_offset] = avg;
        //img_out[shift_offset] = img[offset];
        patch(img,3,row+pad_h2,col+pad_w2+1,i_width,out);
        average(out,9,avg);
        img_out[shift_offset+1] = avg;
        patch(img,3,row+pad_h2+1,col+pad_w2,i_width,out);
        average(out,9,avg);
        img_out[shift_offset+width] = avg;
        patch(img,3,row+pad_h2+1,col+pad_w2+1,i_width,out);
        average(out,9,avg);
        img_out[shift_offset+width+1] = avg;*/
    }    
}
