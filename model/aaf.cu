#include "hip/hip_runtime.h"
extern "C" __global__
void filter_5x5(const short* source_image, int row, int col,const double* filter,int width,short &pix_out) {
    int offset = 5/2;
    float average = 0.;
    float weight = 0.;
    int shift_offset = (row)*width + col ;
    for(int i=0;i<5;i++) {
        for(int j=0;j<5;j++) {
            int avg;
            int y = row + i - offset;
            int x = col + j - offset;
            average = average + filter[i*5+j]*source_image[y*width+x];  
            weight = weight + filter[i*5+j];
        }
    }
    //pix_out = source_image[shift_offset];
    pix_out = int(average/weight);
}

extern "C" __global__
void aaf(const short* img,int width, int height,int pad_w,int pad_h,const double* filter,short* img_out) {
  
    int row = (blockIdx.y * blockDim.y + threadIdx.y);
    int col = (blockIdx.x * blockDim.x + threadIdx.x);
    int pad_w2 = pad_w/2;
    int pad_h2 = pad_h/2;
    int i_width = width + pad_w;
    int i_height = height + pad_h;
    short pix_out;
    //if ((row < height-pad_h2 && row>=pad_h2) && (col < width-pad_w2 && col>=pad_w2)) {
        int offset = row*width + col ;
        int shift_offset = (row+pad_h2)*i_width + col + pad_w2;
        //img_out[offset] = img[shift_offset];
        filter_5x5(img,row+pad_h2,col+pad_w2,filter,i_width,pix_out);
        img_out[offset] = pix_out;
    //}    
}
