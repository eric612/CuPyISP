
#include <hip/hip_runtime.h>
extern "C" __global__
void gac(const short* img,int width, int height,const short* lut,int divide,short* img_out) {
  
    int row = (blockIdx.y * blockDim.y + threadIdx.y);
    int col = (blockIdx.x * blockDim.x + threadIdx.x);
    
    if ((row < height && row>=0) && (col < width && col>=0)) {
        int offset = row*width*3 + col*3 ;
        
        img_out[offset] =  lut[img[offset]]/divide;
        img_out[offset+1] =  lut[img[offset+1]]/divide;
        img_out[offset+2] =  lut[img[offset+2]]/divide;
        
    }    
}
extern "C" __global__
void slut(unsigned short* lut,double gamma,int maxval) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    //lut[index] = round(pow(float(index)/float(maxval), gamma)* maxval);
    //lut[index+1] = round(pow(float(index+1)/float(maxval), gamma)* maxval);
    //lut[index+2] = round(pow(float(index+2)/float(maxval), gamma)* maxval);
    //lut[index+3] = round(pow(float(index+3)/float(maxval), gamma)* maxval);
    lut[index] = round(pow(float(index)/float(maxval), float(gamma))* maxval);

}