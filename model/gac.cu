
#include <hip/hip_runtime.h>
#define BOUND(a,min_val,max_val)           ( (a < min_val) ? min_val : (a >= max_val) ? (max_val) : a )
extern "C" __global__
void gac(const short* img,int width, int height,const short* lut,int divide,int maxval,short* img_out) {
  
    int row = (blockIdx.y * blockDim.y + threadIdx.y);
    int col = (blockIdx.x * blockDim.x + threadIdx.x);
    
    if ((row < height && row>=0) && (col < width && col>=0)) {
        int offset = row*width*3 + col*3 ;
        int pix = BOUND(img[offset],0,maxval);
        img_out[offset] =  lut[pix]/divide;
        pix = BOUND(img[offset+1],0,maxval);
        img_out[offset+1] =  lut[pix]/divide;
        pix = BOUND(img[offset+2],0,maxval);
        img_out[offset+2] =  lut[pix]/divide;
        
    }    
}

extern "C" __global__
void slut(unsigned short* lut,double gamma,int maxval) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    //lut[index] = round(pow(float(index)/float(maxval), gamma)* maxval);
    //lut[index+1] = round(pow(float(index+1)/float(maxval), gamma)* maxval);
    //lut[index+2] = round(pow(float(index+2)/float(maxval), gamma)* maxval);
    //lut[index+3] = round(pow(float(index+3)/float(maxval), gamma)* maxval);
    lut[index] = BOUND(round(pow(float(index)/float(maxval), float(gamma))* maxval),0,maxval);

}